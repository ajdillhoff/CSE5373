
#include <hip/hip_runtime.h>
__global__ void sumReduceKernel(float *input, float *output) {
    unsigned int i = 2 * threadIdx.x;

    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        // Only threads in even positions participate
        if (threadIdx.x % stride == 0) {
            input[i] += input[i + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        *output = input[0];
    }
}

__global__ void sumReduceConvergentKernel(float *input, float *output) {
    unsigned int i = threadIdx.x;

    for (unsigned int stride = blockDim.x; stride >= 1; stride /= 2) {
        if (i < stride) {
            input[i] += input[i + stride];
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        *output = input[0];
    }
}

int sumReduce(float *input, int size) {
    float *d_input, *d_output;
    int output;

    hipMalloc((void **)&d_input, size * sizeof(float));
    hipMalloc((void **)&d_output, sizeof(float));

    hipMemcpy(d_input, input, size * sizeof(float), hipMemcpyHostToDevice);

    unsigned int numActiveThreads = size / 2;

    dim3 dimBlock(numActiveThreads, 1, 1);
    dim3 dimGrid(numActiveThreads / 1024 + 1, 1, 1);

    sumReduceKernel<<<dimGrid, dimBlock>>>(d_input, d_output);
    hipDeviceSynchronize();

    sumReduceConvergentKernel<<<dimGrid, dimBlock>>>(d_input, d_output);
    hipDeviceSynchronize();

    hipMemcpy(&output, d_output, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}